#include "hip/hip_runtime.h"
#include <>
#include <iostream>
#include <cstdio> 
#include <cstdlib>   
#include <hip/hip_runtime.h> 

struct vertice {
    int d;
    int first;
    int indeg;
    int outdeg;
    int lastchangeit;
};

struct edge {
    int s;
    int head;
    int tail;
    int next;
};

int maxlength = 11000000;

__global__ void findedge(struct vertice* v, struct edge* e, int* qv, int* qe,int* numv,int* nume)
{
    int index = threadIdx.x+1+blockIdx.x*blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index;i <= (*numv);i = i + stride)
    {
        int u = qv[i];
        for (int j = v[u].first;j != 0;j = e[j].next)
        {
            if (v[e[j].tail].indeg == 1)
            {
                v[e[j].tail].d = min(v[e[j].tail].d, v[e[j].head].d + e[j].s);
            }
            if (v[e[j].tail].outdeg == 0)
            {
                continue;
            }
            atomicExch(&(qe[atomicAdd(nume, 1) + 1]), j);
        }
    }
}

__global__ void release(struct vertice* v, struct edge* e, int* qv, int* qe, int* numv,int* nume,int *it)
{
    int index = threadIdx.x + 1 + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index;i <= (*nume);i = i + stride)
    {
        //printf("%d ", i);
        int ee = qe[i];
        //printf("%d ", ee);
        if (v[e[ee].tail].d > v[e[ee].head].d + e[ee].s)
        {
            //atomicExch(&(qv[atomicAdd(numv, 1) + 1]), e[ee].tail);
            atomicMin(&(v[e[ee].tail].d), v[e[ee].head].d + e[ee].s);
            v[e[ee].tail].lastchangeit = *it;
            /*if (*it != v[e[ee].tail].lastchangeit)
            {
                (*numv)++;
                qv[*numv] = e[ee].tail;
                
            }
            */
        }
    }
}

__global__ void findvertice(struct vertice* v, struct edge* e, int* qv, int* qe, int* numv, int* nume, int* it,int n)
{
    int index = threadIdx.x + 1 + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index;i <= n;i = i + stride)
    {
        if (v[i].lastchangeit == *it)
        {
            atomicExch(&(qv[atomicAdd(numv, 1) + 1]), i);
        }
    }
}

int main()
{
    int n, m;
    freopen("hbf.txt", "r", stdin);
    scanf("%d %d", &n, &m);
    struct vertice* v;
    struct edge* e;
    int* qv;
    int* qe;
    int* numv;
    int* nume;
    int* it;
    hipMallocManaged((void**)&v, (n + 1) * sizeof(struct vertice));
    hipMallocManaged((void**)&e, (m + 1) * sizeof(struct edge));
    hipMallocManaged((void**)&qv, (n+1) * sizeof(int));
    hipMallocManaged((void**)&qe, (m+1) * sizeof(int));
    hipMallocManaged((void**)&numv, sizeof(int));
    hipMallocManaged((void**)&nume, sizeof(int));
    hipMallocManaged((void**)&it, sizeof(int));
    for (int i = 1;i <= n;i++)
    {
        v[i].d = maxlength;
        v[i].first = 0;
        v[i].indeg = 0;
        v[i].outdeg = 0;
        v[i].lastchangeit = 0;
    }
    v[1].d = 0;
    for (int i = 1;i <= m;i++)
    {
        e[i].head = 0;
        e[i].tail = 0;
        e[i].next = 0;
        e[i].s = 0;
    }
    *numv = 0;
    *nume = 0;
    //qv[1] = 1;
    for (int i = 1;i <= m;i++)
    {
        int p, q, l;
        scanf("%d %d %d", &p, &q, &l);
        if (p != q)
        {
            v[q].indeg++;
            v[p].outdeg++;
            if (p == 1)
            {
                v[q].d= min(v[q].d, l);
                (*numv)++;
                //printf("%d ", p);
                //printf("%d ", *numv);
                //printf("%d ", qv[*numv]);
                //printf("%d -> ", q);
                qv[*numv] = q;
                //printf("%d %d %d\n", *numv, qv[*numv], q);
            }
            e[i].next = v[p].first;
            v[p].first = i;
            e[i].head = p;
            e[i].tail = q;
            e[i].s = l;
        }
        else
        {
            i--;
            m--;
        }
    }
    freopen("CON", "r", stdin);
    for (*it = 1;(*it) <= n-2;(*it)++)
    {
        dim3 blockSize(256);
        dim3 gridSize1(((*numv) + blockSize.x - 1) / blockSize.x);
        findedge << <gridSize1, blockSize >> > (v, e, qv, qe,numv,nume);
        hipDeviceSynchronize();
        /*printf("\ne: ");
        for (int j = 1;j <= (*nume);j++)
        {
            printf("%d %d %d %d\n", qe[j],e[qe[j]].head, e[qe[j]].tail, e[qe[j]].s);
        }
        printf("v: ");*/
        *numv = 0;
        dim3 gridSize2(((*nume) + blockSize.x - 1) / blockSize.x);
        release << <gridSize2, blockSize >> > (v, e, qv, qe, numv,nume,it);
        hipDeviceSynchronize();
        *nume = 0;
        /*for (int j = 1;j <= (*numv);j++)
        {
            printf("%d ", qv[j]);
        }*/
        dim3 gridSize3((n + blockSize.x - 1)/blockSize.x);
        findvertice << <gridSize3, blockSize >> > (v, e, qv, qe, numv, nume, it,n);
        hipDeviceSynchronize();
        /*printf("\nd: ");
        for (int j = 1;j <= n;j++)
        {
            printf("%d ", v[j].d);
        }*/
        if (*numv == 0)
            break;
        //printf("\n");
    }
    for (int i = 1;i <= m;i++)
    {
        if (v[e[i].tail].outdeg == 0)
        {
            v[e[i].tail].d = min(v[e[i].tail].d, v[e[i].head].d + e[i].s);
        }
    }
    freopen("E:\\������\\ʵ����\\bf_no_cuda\\bf_no_cuda\\hbf_result.txt", "w", stdout);
    for (int j = 1;j <= n;j++)
    {
        printf("%d ", v[j].d);
    }
    hipFree(v);
    hipFree(e);
    hipFree(qv);
    hipFree(qe);
    hipFree(numv);
    hipFree(nume);
    fclose(stdin);
    fclose(stdout);
}